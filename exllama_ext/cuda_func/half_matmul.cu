#include "hip/hip_runtime.h"
#include "half_matmul.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#include "../cuda_compat.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

// Block size

const int THREADS_X = 32;     // Block size and thread count along columns in w and out
const int THREADS_Y = 8;      // Block size and thread count along rows in x and out
const int BLOCKSIZE = 256;

__global__ void half_matmul_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    const int height,
    const int dim,
    const int width
)
{
    const int column = (blockIdx.x * THREADS_X + threadIdx.x) * 2;
    const int row = blockIdx.y * THREADS_Y + threadIdx.y;
    const int k0 = blockIdx.z * BLOCKSIZE;

    if (row >= height) return;
    if (column >= width) return;

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    half2* x_ptr = (half2*) x_.item_ptr(row, k0);
    half2* w_ptr = (half2*) w_.item_ptr(k0, column);
    half2 acc = {};

    #pragma unroll
    for (int k = k0; k < k0 + BLOCKSIZE / 2; k++)
    {
        half2 x_item = *x_ptr++;
        half2 x_item_0 = __half2half2(x_item.x);
        half2 x_item_1 = __half2half2(x_item.y);
        half2 w_item_0 = *w_ptr; w_ptr += w_.width / 2;
        half2 w_item_1 = *w_ptr; w_ptr += w_.width / 2;
        acc = __hfma2(x_item_0, w_item_0, acc);
        acc = __hfma2(x_item_1, w_item_1, acc);
    }

    // out_.set(row, column, acc);
    atomicAdd((half2*)out_.item_ptr(row, column), acc);
}

void half_matmul_cuda
(
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    hipStream_t alt_stream
)
{
    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (width + THREADS_X - 1) / THREADS_X / 2,
        (height + THREADS_Y - 1) / THREADS_Y,
        (dim + BLOCKSIZE - 1) / BLOCKSIZE
    );

    half_matmul_kernel<<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width);
}

// cuBLAS can't be beat for large matrices, probably

void half_matmul_cublas_cuda
(
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    hipblasHandle_t handle,
    bool no_zero,
    hipStream_t alt_stream
)
{
    // Fall back on a naive kernel for small matmuls to avoid cuBLAS overhead

    if (height < 4 && dim <= 8192)
    {
        half_matmul_small_cuda(x, w, out, height, dim, width, no_zero, alt_stream);
        return;
    }

    // Use cuBLAS

    const half alpha = __float2half(1.0f);
    const half beta = no_zero ? __float2half(1.0f) : __float2half(0.0f);

    hipStream_t default_stream;
    if (alt_stream)
    {
        hipblasGetStream(handle, &default_stream);
        hipblasSetStream(handle, alt_stream);
    }

    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, w, width, x, dim, &beta, out, width);

    if (alt_stream)
    {
        hipblasSetStream(handle, default_stream);
    }
}

// Alternative to cuBLAS for tall or wide matrices

const int S_THREADS_X = 8;     // width
const int S_THREADS_Y = 1;     // height
const int S_BLOCKSIZE = 64;    // dim/k
const int MAX_DIM = 8192;

template<bool use_half2>
__global__ void half_matmul_small_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    const int height,
    const int dim,
    const int width,
    bool no_zero
)
{
    const int column = blockIdx.x * S_THREADS_X + threadIdx.x;
    const int row = blockIdx.y * S_THREADS_Y + threadIdx.y;
    const int k = threadIdx.z * S_BLOCKSIZE;

    if (row >= height) return;
    if (column >= width) return;
    //if (k >= dim) return;

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    int k_end = k + S_BLOCKSIZE;
    if (k_end > dim) k_end = dim;

    const half* x_ptr = x_.item_ptr(row, k);
    const half* x_ptr_end = x_.item_ptr(row, k_end);

    if constexpr(use_half2)
    {
        const half2* w_ptr = (half2*) w_.item_ptr(k, column * 2);
        half2* out_ptr = (half2*) out_.item_ptr(row, column * 2);
        half2 r = {};

        while(x_ptr < x_ptr_end)
        {
            #pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                half x_item = *x_ptr++;
                half2 x_items = __half2half2(x_item);
                half2 w_items = *w_ptr; w_ptr += width / 2;
                r = __hfma2(x_items, w_items, r);
            }
        }

        __shared__ half2 accum[MAX_DIM / S_BLOCKSIZE][S_THREADS_X];
        accum[threadIdx.z][threadIdx.x] = r;

        __syncthreads();
        if (threadIdx.z == 0)
        {
            half2 acc = accum[0][threadIdx.x];
            for (int i = 1; i < gridDim.z; ++i) acc = __hadd2(acc, accum[i][threadIdx.x]);
            if (no_zero) acc = __hadd2(acc, *out_ptr);
            *out_ptr = acc;
        }
    }
    else
    {
        const half* w_ptr = w_.item_ptr(k, column);
        half r = {};

        while(x_ptr < x_ptr_end)
        {
            #pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                half x_item = *x_ptr++;
                half w_item = *w_ptr; w_ptr += width;
                r = __hfma(x_item, w_item, r);
            }
        }

        __shared__ half accum[MAX_DIM / S_BLOCKSIZE][S_THREADS_X];
        accum[threadIdx.z][threadIdx.x] = r;

        __syncthreads();
        if (threadIdx.z == 0)
        {
            half acc = accum[0][threadIdx.x];
            for (int i = 1; i < gridDim.z; ++i) acc = __hadd(acc, accum[i][threadIdx.x]);
            if (no_zero) acc = __hadd(acc, out_.item(row, column));
            out_.set(row, column, acc);
        }
    }
}

void half_matmul_small_cuda
(
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    bool no_zero,
    hipStream_t alt_stream
)
{
    bool use_half2 = true;

    dim3 threads
    (
        S_THREADS_X,
        S_THREADS_Y,
        (dim + S_BLOCKSIZE - 1) / S_BLOCKSIZE
    );

    dim3 blocks
    (
        (width + S_THREADS_X - 1) / S_THREADS_X / (use_half2 ? 2 : 1),
        (height + S_THREADS_Y - 1) / S_THREADS_Y,
        1
    );

    half_matmul_small_kernel<true><<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width, no_zero);
}

