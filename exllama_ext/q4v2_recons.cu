#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdint>

// Block size

const int THREADS_X = 64;     // Block size and thread count along columns in out, each thread converts 2 columns
const int THREADS_Y = 4;       // Block size and thread count along rows in x and out, each thread converts BLOCK_SIZE_Y * 8 rows
const int BLOCK_SIZE_Y = 1;    // * 8 rows

__global__ void q4v2_recons_kernel
(
    const uint32_t* w,
    half* out,  // (y)
    const half* w_scales,
    const uint32_t* w_zeros,
    const int height,
    const int width,
    const int groupsize
)
{
    // Start of input

    uint64_t* w2 = (uint64_t*) w;
    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    int w2_row = (THREADS_Y * blockIdx.y + threadIdx.y) * BLOCK_SIZE_Y;
    int w2_stride = width >> 1;

    // Start of output

    half2* out2 = (half2*) out;
    int out2_column = w2_column;
    int out2_row = w2_row * 8;
    int out2_stride = width >> 1;

    // Group, zeros and scales

    int groupsize8 = groupsize >> 3;
    int group_idx = w2_row / groupsize8;
    int next_group = group_idx * groupsize8;  // first iteration will advance to first group

    half2* w_scales2 = (half2*) w_scales;
    int w_scales2_column = w2_column;
    int w_scales2_stride = width >> 1;

    uint64_t* w_zeros2 = (uint64_t*) w_zeros;
    int w_zeros2_column = w2_column >> 3;           // w_column >> 4
    int w_zeros2_shift = (w2_column & 0x07) << 3;   // shift 2*4 bits per double column
    int w_zeros2_stride = (width >> 4);

    // Indices

    int w2_idx = w2_row * w2_stride + w2_column;
    int w_scales2_idx = group_idx * w_scales2_stride + w_scales2_column;
    int w_zeros2_idx = group_idx * w_zeros2_stride + w_zeros2_column;

    int out2_idx = out2_row * out2_stride + out2_column;

    // Loop over BLOCK_SIZE_Y

    //int w2_row_end = min(w2_row + BLOCK_SIZE_Y, height);
    int w2_row_end = w2_row + BLOCK_SIZE_Y;

    half2 w_scale2;
    int w_zerol_q, w_zeror_q;

    while (w2_row < w2_row_end)
    {
        if (BLOCK_SIZE_Y == 1 || w2_row >= next_group)  // optimizer should remove this if BLOCK_SIZE_Y == 1
        {
            w_scale2 = w_scales2[w_scales2_idx];

            uint64_t w_zero2_packed = w_zeros2[w_zeros2_idx];
            int w_zero2_q = ((w_zero2_packed >> w_zeros2_shift) & 0xff) + 0x11;
            w_zerol_q = w_zero2_q & 0x0f;
            w_zeror_q = w_zero2_q >> 4;

            w_scales2_idx += w_scales2_stride;
            w_zeros2_idx += w_zeros2_stride;
            next_group += groupsize8;
        }

        // Read 2 * 8 packed quants from w2

        uint64_t w2_read = w2[w2_idx];
        w2_idx += w2_stride;

        // Convert quants to half2

        half2 w2_0 = __halves2half2(__int2half_rn((int)((w2_read >>  0) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 32) & 0x0f) - w_zeror_q));
        half2 w2_1 = __halves2half2(__int2half_rn((int)((w2_read >>  4) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 36) & 0x0f) - w_zeror_q));
        half2 w2_2 = __halves2half2(__int2half_rn((int)((w2_read >>  8) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 40) & 0x0f) - w_zeror_q));
        half2 w2_3 = __halves2half2(__int2half_rn((int)((w2_read >> 12) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 44) & 0x0f) - w_zeror_q));
        half2 w2_4 = __halves2half2(__int2half_rn((int)((w2_read >> 16) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 48) & 0x0f) - w_zeror_q));
        half2 w2_5 = __halves2half2(__int2half_rn((int)((w2_read >> 20) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 52) & 0x0f) - w_zeror_q));
        half2 w2_6 = __halves2half2(__int2half_rn((int)((w2_read >> 24) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 56) & 0x0f) - w_zeror_q));
        half2 w2_7 = __halves2half2(__int2half_rn((int)((w2_read >> 28) & 0x0f) - w_zerol_q), __int2half_rn((int)((w2_read >> 60) & 0x0f) - w_zeror_q));

        out2[out2_idx] = __hmul2(w2_0, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_1, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_2, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_3, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_4, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_5, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_6, w_scale2); out2_idx += out2_stride;
        out2[out2_idx] = __hmul2(w2_7, w_scale2); out2_idx += out2_stride;

        w2_row++;
    }
}


// Convert w -> y, from q4 to half
//
// Shape of w is [height, width], dtype = q4
// Output shape is [height, width], dtyle = half
// Shape of w_scales is [height / groupsize, width], dtype = 4-bit quant (packed rows)
// Shape of w_zeros is [height / groupsize, width], dtype = half

void q4v2_recons_cuda
(
    const uint32_t* w,
    half* out,  // y
    const half* w_scales,
    const uint32_t* w_zeros,
    const int height,
    const int width,
    const int groupsize
)
{
    dim3 threads
    (
        THREADS_X,
        THREADS_Y,
        1
    );

    dim3 blocks
    (
        (width + threads.x - 1) / threads.x / 2,
        (height + threads.y - 1) / threads.y / BLOCK_SIZE_Y,
        1
    );

    q4v2_recons_kernel<<<blocks, threads>>>
    (
        w,
        out,
        w_scales,
        w_zeros,
        height,
        width,
        groupsize
    );
}

