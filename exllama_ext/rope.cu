#include "hip/hip_runtime.h"
#include "rope.h"
#include "util.h"
#include "matrix.h"

const int THREADS_X = 32;
const int THREADS_Y = 4;
const int MAX_POS_EMBEDDINGS = 32768;  // Actual number doesn't matter

__global__ void rope_cuda_kernel
(
    half* x,
    const half* sin,
    const half* cos,
    int rows,
    int head_dim,
    int num_heads,
    int past_len
)
{
    MatrixView_half_rw x_(x, rows, head_dim);
    MatrixView_half sin_(sin, MAX_POS_EMBEDDINGS, head_dim);
    MatrixView_half cos_(cos, MAX_POS_EMBEDDINGS, head_dim);

    // Assume head_dim is a power of two (it's always 128 for Llama)

    int column = (blockIdx.x * THREADS_X + threadIdx.x) * 2;
    int row = blockIdx.y * THREADS_Y + threadIdx.y;
    if (row >= rows) return;

    // Get sin and cos

    int sincos_row = past_len + row / num_heads;
    int half_dim = head_dim / 2;

    half2 cos2_l = cos_.item_half2(sincos_row, column);
    half2 cos2_r = cos_.item_half2(sincos_row, column + half_dim);
    half2 sin2_l = sin_.item_half2(sincos_row, column);
    half2 sin2_r = sin_.item_half2(sincos_row, column + half_dim);
    sin2_l = __hneg2(sin2_l);

    // Apply embedding to num_heads rows

    //#pragma unroll
    //for (int k = row; k < row + num_heads; k++)
    int k = row;
    {
//         if (k == 0)
//         {
//             printf("%f %f %f %f  - ", __half2float(sin2_l.x), __half2float(sin2_l.y), __half2float(sin2_r.x), __half2float(sin2_r.y));
//             printf("%f %f %f %f \n", __half2float(cos2_l.x), __half2float(cos2_l.y), __half2float(cos2_r.x), __half2float(cos2_r.y));
//         }

        half2 item2_l = x_.item_half2(k, column);
        half2 item2_r = x_.item_half2(k, column + half_dim);
        half2 item2_ls = __hmul2(item2_r, sin2_l);
        half2 item2_rs = __hmul2(item2_l, sin2_r);
        item2_l = __hfma2(item2_l, cos2_l, item2_ls);
        item2_r = __hfma2(item2_r, cos2_r, item2_rs);
//         item2_l = __hmul2(item2_l, cos2_l);
//         item2_r = __hmul2(item2_r, cos2_r);
//         item2_l = __hadd2(item2_l, item2_ls);
//         item2_r = __hadd2(item2_r, item2_rs);
        x_.set_half2(k, column, item2_l);
        x_.set_half2(k, column + half_dim, item2_r);
//        x_.set_half2(k, column, __half2half2(__int2half_rn(69)));
//        x_.set_half2(k, column + half_dim, __half2half2(__int2half_rn(69)));

    }
}

hipError_t rope_cuda
(
    half* x,
    const half* sin,
    const half* cos,
    const int rows,
    const int head_dim,
    const int num_heads,
    const int past_len
)
{

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (head_dim + THREADS_X - 1) / THREADS_X / 2 / 2,
        (rows + THREADS_Y - 1) / THREADS_Y, //(rows + num_heads - 1) / num_heads,
        1
    );

    rope_cuda_kernel<<<blocks, threads>>>(x, sin, cos, rows, head_dim, num_heads, past_len);

}
